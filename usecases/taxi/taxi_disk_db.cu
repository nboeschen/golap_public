#include <iostream>
#include <hip/hip_runtime_api.h>
#include <gflags/gflags.h>

#include "hl/serialization.hpp"
#include "taxi.hpp"

DEFINE_uint32(scale_factor, 10, "Scale factor.");
DEFINE_uint32(all_peek, 0, "Number of lines of all tables to peek at.");
DEFINE_string(path, "../bin/disk.dat", "Path to output file.");
DEFINE_string(op, "read", "Either read,write,bin2csv,csv2bin");
DEFINE_string(format, "binary", "binary or csv");
DEFINE_string(csv_delimiter, ";", "CSV delimiter");


int main(int argc, char *argv[]){
    gflags::ParseCommandLineFlags(&argc, &argv, true);

    // golap::StorageManager sm{FLAGS_path};
    TaxiVar var{(uint32_t)-1, (uint32_t)-1, "", "", FLAGS_scale_factor};

    TaxiColLayout db(var,(FLAGS_op == "write" ? "init_populate" : "init_only"));

    if (FLAGS_op == "csv2bin"){
        golap::read_col_db_csv(db.tables,FLAGS_path,FLAGS_csv_delimiter);
        util::Log::get().info_fmt("Read csv table data.");
        FLAGS_path += ".dat";
        golap::write_col_db_bin(db.tables,FLAGS_path);
    }else if (FLAGS_op == "bin2csv"){
        golap::read_col_db_bin(db.tables,FLAGS_path);
        util::Log::get().info_fmt("Read dat table data.");
        FLAGS_path += ".csv";
        golap::write_col_db_csv(db.tables,FLAGS_path,FLAGS_csv_delimiter);
    }else if(FLAGS_format == "binary"){
        if(FLAGS_op == "read"){
            golap::read_col_db_bin(db.tables,FLAGS_path);
        }else if(FLAGS_op == "write"){
            golap::write_col_db_bin(db.tables,FLAGS_path);
        }
    }else if (FLAGS_format == "csv"){
        if(FLAGS_op == "read"){
            golap::read_col_db_csv(db.tables,FLAGS_path,FLAGS_csv_delimiter);
        }else if(FLAGS_op == "write"){
            golap::write_col_db_csv(db.tables,FLAGS_path,FLAGS_csv_delimiter);
        }
    }


    if(FLAGS_all_peek != 0){
        db.tables.apply([&](auto& table){
            table.to_csv(std::cout, FLAGS_csv_delimiter, 0, FLAGS_all_peek);
        });
    }

    return 0;
}

